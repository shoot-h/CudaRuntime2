#include "hip/hip_runtime.h"
﻿//CUDAプログラミングによる差分法を用いたラプラス方程式の解くプログラム
//並列化により，計算時間がどれだけ短縮するかを計測する
//初期条件はすでに与えられているものとする
//参考：https://www2.akita-nct.ac.jp/saka/Lecturenote/lecture/5e/text/26,27.pdf

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

# define hSize 2048

//差分法によるラプラス方程式計算
__global__ void Laplace(float* u,int* flag)
{
    int i = threadIdx.x;
    for (int j = 0; j < 10; j++) {
        if (flag[i] == 0) {
            u[i] = 0.25 * (u[i + 1] + u[i - 1] + u[i + hSize] + u[i - hSize]);
        }
    }
} 

//初期化関数
void Initialize(float* u,int* flag) 
{
    //境界条件設定
    for (int i = 0; i < hSize; i++) {
        flag[i * hSize] = 1;
        flag[(i * hSize) + hSize - 1] = 1;
        flag[i] = 1;
        flag[i + (hSize - 1) * hSize] = 1;
        u[i * hSize] = 0;
        u[(i * hSize) + hSize - 1] = 0;
        u[i] = 0;
        u[i + (hSize - 1) * hSize] = 0;
    }

    //初期ポテンシャル決定
    for (int i = 1; i < hSize - 1; i++) {
        for (int j = 1; j < hSize - 1; j++) {
            if (((i - (0.25 * hSize)) * (i - (0.25 * hSize))) + ((j - (0.75 * hSize)) * (j - (0.75 * hSize))) <= 0.125 * hSize * 0.125 * hSize) {
                flag[i + (j * hSize)] = 1;
                u[i + (j * hSize)] = 100;
            }
            else if (((i - (0.875 * hSize)) * (i - (0.875 * hSize))) + ((j - (0.125 * hSize)) * (j - (0.125 * hSize))) <= 0.05 * hSize * 0.05 * hSize) {
                flag[i + (j * hSize)] = 1;
                u[i + (j * hSize)] = 20;
            }
            else {
                flag[i + (j * hSize)] = 0;
                u[i + (j * hSize)] = 0;
            }

        }
    }
}

//結果出力関数
void OutputResult(float calcTime,float* u) 
{
    //計算時間出力
    printf("%f\n", calcTime);

    /*for (int i = 0; i < h; i++) {
        for (int j = h - 1; j >= 0; j--) {
            printf("%f\t", u[i + (j * h)]);
        }
        printf("\n");
    } //計算できているか確認用*/
}

int main(int argc,char* argv[])
{
    float u[hSize*hSize]; //ポテンシャル保存用配列
    int flag[hSize*hSize]; //固定ポテンシャルかを判別する配列(固定：1，可変：0)

    //計算時間計測用
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float calcTime = 0;

    //GPU上にメモリ確保
    float* d_u;
    int* d_flag;
    size_t floatsize = hSize * hSize * sizeof(float);
    size_t intsize = hSize * hSize * sizeof(int);
    hipMalloc((void**)&d_u, floatsize);
    hipMalloc((void**)&d_flag, floatsize);

    Initialize(u,flag);//初期化

    //GPUメモリにデータコピー
    hipMemcpy(d_flag, flag, intsize, hipMemcpyHostToDevice);
    hipMemcpy(d_u, u, floatsize, hipMemcpyHostToDevice);

    //グリッドとブロック指定
    const int threadsPerGrid = 1;
    const int threadsPerBlock = hSize*hSize;

    //ラプラス方程式計算計算
    hipEventRecord(start);
    Laplace << <threadsPerGrid, threadsPerBlock >> > (d_u,d_flag);
    hipMemcpy(u, d_u, floatsize, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&calcTime, start, stop);//計算時間算出

    OutputResult(calcTime,u);//結果出力

    //メモリ開放
    hipEventDestroy(start);hipEventDestroy(stop);hipFree(d_u);hipFree(d_flag);

    return 0;
}
